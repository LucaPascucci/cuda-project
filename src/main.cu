#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>

using namespace std;

#define LOCAL_N 1024
#define RADIUS 3
#define BLOCK_SIZE 16
#define N (2048*2048)
#define THREAD_PER_BLOCK 512

__global__ void mykernel(void) { //funzione eseguita nel device e chiamata dal codice principale

}

__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
}

__global__ void vectorAddBlocks(int *a, int *b, int *c) {	//blockIdx.x utilizzato come indice dell'array ogni block geestisce un elemento differente dell'array
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

__global__ void vectorAddThreads(int *a, int *b, int *c) {
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

__global__ void vectorAddBlocksThreads(int *a, int *b, int *c){ //blockDim corrisponde alla grandezza del blocco
	int index = threadIdx.x + blockIdx.x * blockDim.x; //serve per ricavare l'effettivo in indice del'array
	//es 5(indice trhead) + 2(indice blocco) * 8 (Dim di ogni blocco) = 21 indice nell'array)
	c[index] = a[index] + b[index];
}

__global__ void stencil_1d(int *in, int *out) {
	__shared__ int temp[BLOCK_SIZE + 2 * RADIUS]; //dati condivisi e visibili solo tra i threads in uno stesso block
	int gindex = threadIdx.x + blockIdx.x * blockDim.x; //serve per ricavare l'effettivo in indice del'array
	int lindex = threadIdx.x + RADIUS; //indice del vettore temp dove scrivere i dati
	// Read input elements into shared memory
	temp[lindex] = in[gindex];
	if (threadIdx.x < RADIUS) {
		temp[lindex - RADIUS] = in[gindex - RADIUS];
		temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
	}
	// Synchronize (ensure all the data is available) sincronizza tutti i thread di un blocco
	//Usato per prevenire RAW,WAR,WAW hazards
	__syncthreads(); //Come fosse una barriera che tutti i thread devono raggiungere assieme
	// Apply the stencil
	int result = 0;
	for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
		result += temp[lindex + offset];

	// Store the result
	out[gindex] = result;
}

void random_ints (int* a, int n){
	for (int i = 0; i < n; ++i){
		a[i] = rand();
	}
}

void vectorAddBlocksThreads(){
	int *a,*b,*c;				// copie di a b c presenti nel pc
	int *d_a, *d_b, *d_c;		// copie di a b c presenti nella scheda grafica
	int size = N * sizeof(int);

	//alloca spazio per a b c nella scheda grafica
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);

	//Alloco memoria per l'host
	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	//inizializzo i vettori
	random_ints(a,N);
	random_ints(b,N);

	//copia i valori nella scheda grafica
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	//lancia add() kernel sulla GPU con N threads
	vectorAddBlocksThreads<<<N/THREAD_PER_BLOCK,THREAD_PER_BLOCK>>>(d_a,d_b,d_c);

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	free(a);
	free(b);
	free(c);

	//Libera memoria sulla GPU
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

}

void vectorAddThreads(){
	int *a,*b,*c;				// copie di a b c presenti nel pc
	int *d_a, *d_b, *d_c;		// copie di a b c presenti nella scheda grafica
	int size = N * sizeof(int);

	//alloca spazio per a b c nella scheda grafica
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);

	//Alloco memoria per l'host
	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	//inizializzo i vettori
	random_ints(a,N);
	random_ints(b,N);

	//copia i valori nella scheda grafica
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	//lancia add() kernel sulla GPU con N threads
	vectorAddThreads<<<1,N>>>(d_a,d_b,d_c);

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	free(a);
	free(b);
	free(c);

	//Libera memoria sulla GPU
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

}

void vectorAddBlocks(){
	int *a,*b,*c;				// copie di a b c presenti nel pc
	int *d_a, *d_b, *d_c;		// copie di a b c presenti nella scheda grafica
	int size = N * sizeof(int);

	//alloca spazio per a b c nella scheda grafica
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);

	//Alloco memoria per l'host
	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	//inizializzo i vettori
	random_ints(a,N);
	random_ints(b,N);

	//copia i valori nella scheda grafica
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	//lancia add() kernel sulla GPU con N blocks
	vectorAddBlocks<<<N,1>>>(d_a,d_b,d_c);

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	free(a);
	free(b);
	free(c);

	//Libera memoria sulla GPU
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

void simpleAdd(){

	int a,b,c;				// copie di a b c presenti nel pc
	int *d_a, *d_b, *d_c;	// copie di a b c presenti nella scheda grafica
	int size = sizeof(int);

	//alloca spazio per a b c nella scheda grafica
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);

	a = 2;
	b = 7;

	//copia i valori nella scheda grafica
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	//lancia add() kernel sulla GPU
	add<<<1,1>>>(d_a,d_b,d_c);

	// Copy result back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	//Libera memoria sulla GPU
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	printf("La somma fa: %d\n",c);
}

void stencil(){
	int *in, *out;			// host copies of in, out
	int *d_in, *d_out;		// device copies of in, out
	int size = (LOCAL_N + 2*RADIUS) * sizeof(int); //dimensioni degli array

	// Alloc space for host copies and setup values
	in = (int *)malloc(size);
	random_ints(in, size);
	out = (int *)malloc(size);

	// Alloc space for device copies
	hipMalloc((void **)&d_in, size);
	hipMalloc((void **)&d_out, size);

	// Copy to device
	hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
	hipMemcpy(d_out, out, size, hipMemcpyHostToDevice);

	// Launch stencil_1d() kernel on GPU
	stencil_1d<<<LOCAL_N/BLOCK_SIZE,BLOCK_SIZE>>>(d_in + RADIUS, d_out + RADIUS);
	// Copy result back to host
	hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

	// Cleanup
	free(in);
	free(out);
	hipFree(d_in);
	hipFree(d_out);
}

int main(void) {

	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf(" Device name: %s\n", prop.name);
		printf(" Total Global Memory: %zd\n", prop.totalGlobalMem);
		printf(" Shared Memory Per Block: %zd\n", prop.sharedMemPerBlock);
		printf(" Memory Clock Rate (KHz): %d\n",prop.memoryClockRate);
		printf(" Memory Bus Width (bits): %d\n",prop.memoryBusWidth);
		printf(" Peak Memory Bandwidth (GB/s): %f\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth/8)/1.0e6);
		printf(" Max Threads Per Block: %d\n", prop.maxThreadsPerBlock);
		printf(" MultiProcessorCount: %d\n\n", prop.multiProcessorCount);
	}

	clock_t t1,t2;
	float diff;
	t1 = clock();
	//Chiamata dal host code al device code detto anche kernel launch
	mykernel<<<1,1>>>();
	printf("Hello world!\n");
	t2 = clock();
	diff = ((float)(t2 - t1) / 1000000.0F ) * 1000;
	printf("Tempo %f\n",diff);

	t1 = clock();
	simpleAdd();
	t2 = clock();
	diff = ((float)(t2 - t1) / 1000000.0F ) * 1000;
	printf("Tempo simpleAdd: %f\n",diff);

	t1 = clock();
	vectorAddBlocks();
	t2 = clock();
	diff = ((float)(t2 - t1) / 1000000.0F ) * 1000;
	printf("Tempo vectorAddBlocks: %f\n",diff);

	t1 = clock();
	vectorAddThreads();
	t2 = clock();
	diff = ((float)(t2 - t1) / 1000000.0F ) * 1000;
	printf("Tempo vectorAddThreads: %f\n",diff);

	t1 = clock();
	vectorAddBlocksThreads();
	t2 = clock();
	diff = ((float)(t2 - t1) / 1000000.0F ) * 1000;
	printf("Tempo vectorAddBlocksThreads: %f\n",diff);

	t1 = clock();
	stencil();
	t2 = clock();
	diff = ((float)(t2 - t1) / 1000000.0F ) * 1000;
	printf("Tempo stencil: %f\n",diff);

	return 0;
}

